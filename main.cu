/*
 * Russell Taylor(rtaylor)
 * Matt Crusse(macrusse)
 * CPE458-01 Lab 1 Winter 2013 
 */


#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <sys/mman.h> 
#include <errno.h>
#include <string.h>
#include <stdarg.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <stdint.h>

/*Compile-Time Declaration on double or float usage*/
#ifdef DOUBLE
#define TYPEUSE double

#else
#define TYPEUSE float

#endif

/* 
 * Handles CUDA errors, taking from provided sample code on clupo site
 */
/*
static void HandleError( cudaError_t err, const char * file, int line)
{
  if(err !=cudaSuccess){
    printf("%s in %s at line %d\n", cudaGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
*/


/*Reads Input File and Returns Buffer of Contents*/
char* read_file(const char * file_name) 
{
  size_t size;
  char *buffer;
  FILE *fp;
  
  fp = fopen(file_name,"r");
  if(!fp) {
    fprintf (stderr, "Error opening input file.\n");
    exit (EXIT_FAILURE);    
  }

  fseek (fp, 0, SEEK_END);
  size = ftell(fp);
  rewind (fp);
  
  buffer = (char*) malloc (sizeof(char)*size);
  fread (buffer, 1, size, fp);
  fclose(fp);
  return buffer;
}

/*Calculate the Resultant Matrix from Multiplication*/
void calc_matrix(TYPEUSE *A, TYPEUSE *B, TYPEUSE *C, int Arow, int Acol, int Brow, int Bcol)
{
  uint64_t i, j, k;
  TYPEUSE sum = 0;
  for(i = 0; i < Arow; i++)//Iterate through Matrix B columnwise
  {
    for(j = 0; j < Bcol; j++)//Iterate through Matrix A rowwise
    {
        for(k = 0; k < Acol; k++)//Acol = Brow on valid Matrices
        {
          if(i >475)
            printf("");
          sum+= A[ i* (Acol) + k] * B[k * (Bcol) + j];
          
        }
        C[i *Acol + j] = sum;
        sum = 0;
    }
  }
}

/* Print matrix values to a file outputfile */
void output_matrix(const char * outputfile, TYPEUSE *matrix, int row, int col) 
{
  int i, j;

  FILE *ofp = fopen(outputfile, "w");
  if(!ofp){
    fprintf (stderr, "Error opening output file.\n");
    exit (EXIT_FAILURE);    
  }

  for(i = 0; i < row; i++) {
    for(j = 0; j < col; j++) {
      fprintf(ofp, "%.2f ",matrix[i*uint64_t(col) + j]);
    }  
    if(i < row-1){
      fprintf(ofp, "\n");
    }
  }
  fclose(ofp);
}


/*
 * Simply prints out the matrix to screen 
 */
void print_matrix(TYPEUSE *matrix, int row, int col) 
{
  int i, j;
  for(i = 0; i < row; i++) {
    for(j = 0; j < col; j++) {
      //printf("(%d,%d)", i, j);
      printf("%.2f ",matrix[i*col +j]);
    }  
    if(i < row-1){
      printf("\n");
    }
  }
  printf("\n");

}

/*Created a Matrix based on Buffered Input Information*/
TYPEUSE * read_matrix(int * rowCnt, int * colCnt, char * mapped)
{
  TYPEUSE value;  
  const char *delim_space = " ";
  char *token = NULL;  
  char *unconverted;
  int i, j, len;
  TYPEUSE *matrix;
  uint64_t bigiter;
  *colCnt = 0;
  *rowCnt = 0;

  
  /* Determine Col Count */
  i = 0;
  while(mapped[i] != '\n'){
    if(mapped[i] == '.') {
     (*colCnt)++;
    }
    i++;
  }  

  /* Determine Row Count */
  bigiter = 0;//For large file sizes, an int is too small to iterate through
  len = strlen(mapped);
  while(bigiter < len && mapped[bigiter] != '\0'){
    if((mapped[bigiter] == '\n') && (mapped[bigiter+1] != '\0') ) {
     (*rowCnt)++;
    }
    bigiter+=1;
  }
  (*rowCnt)++;

  /* Malloc the Matrix */
  if (( matrix = (TYPEUSE *) malloc((*rowCnt) * (*colCnt) * sizeof(TYPEUSE))) == NULL ) {
    printf("malloc issue");
  }
    
  /* Read values into matrix */
  i = 0; j = 0;
  for (token = strtok(mapped, delim_space); token != NULL; token = strtok(NULL, delim_space)) {
    value = strtod(token, &unconverted);
    matrix[i*(*colCnt) +j] = value;
    j++;
    if(j == (*colCnt)) {
      j = 0;
      if(++i == (*rowCnt))
	      break;
    }
  }
  return matrix;

}

__global__ void MMKernel(TYPEUSE *A_d, TYPEUSE *B_d, TYPEUSE * C_d, int depth, int Awidth, int Bwidth)
{
  TYPEUSE Cvalue = 0.0;

  //int resultLength = Awidth * Bwidth;
  int resultWidth = Bwidth;
  int resultCol = blockIdx.x * blockDim.x + threadIdx.x;
  int resultRow = blockIdx.y * blockDim.y + threadIdx.y;  
  int resultIndex = resultRow * resultWidth + resultCol;

  if(resultRow > Awidth || resultCol > Bwidth)
    return;
    
  for(int k = 0; k < depth; k++) {
    TYPEUSE Aelement = A_d[resultRow * Awidth + k];
    TYPEUSE Belement = B_d[Bwidth * k + resultCol];
    Cvalue += Aelement * Belement;
  }
  C_d[resultIndex] = Cvalue;
}

int main (int argc, const char * argv[])
{
  const char * Cfile = "result.out";
  TYPEUSE * Amatrix, * Bmatrix, * Cmatrix;
  TYPEUSE * A_d, * B_d, * C_d;
  int Arow, Acol, Brow, Bcol;
  int size;
  int blockRow, blockCol;
  char * Amapped, * Bmapped;

  if(argc != 3) { 
    fprintf(stderr, "Usage: [Matrix A] [Matrix B]\n");
    exit(EXIT_FAILURE);
  }

  /* Device Properties */
  /*
  cudaDeviceProp prop;
  cudaGetDeviceProperties(&prop,0);
  printf("maxThreads: %d\n", prop.maxThreadsPerBlock);
  */

  /* Read and Map matrix */
  Amapped = read_file(argv[1]);
  Bmapped = read_file(argv[2]);
  Amatrix = read_matrix(&Arow, &Acol, Amapped); 
  Bmatrix = read_matrix(&Brow, &Bcol, Bmapped);
  if(Acol != Brow) {
    fprintf(stderr, "Matrices are not a compatible size to be multiplied\n");
    exit(EXIT_FAILURE);
  }
  
  /* Malloc a New Matrix */
  if (( Cmatrix = (TYPEUSE *) malloc((Arow) * (Bcol) * sizeof(TYPEUSE))) == NULL ) {
    printf("malloc issue");
  }
  
  /* Malloc and Copy space on GPU */
  size = Arow * Acol * sizeof(TYPEUSE);
  hipMalloc(&A_d, size);
  hipMemcpy(A_d, Amatrix, size, hipMemcpyHostToDevice);
  
  size = Brow * Bcol * sizeof(TYPEUSE);
  hipMalloc(&B_d, size);
  hipMemcpy(B_d, Bmatrix, size, hipMemcpyHostToDevice);

  size = Arow * Bcol * sizeof(TYPEUSE);
  hipMalloc(&C_d, size);
  
  blockRow = (Arow+31) / 32;
  blockCol = (Bcol+31) / 32;
    
  /*Kernel Call*/
  dim3 dimGrid(blockCol,blockRow);
  dim3 dimBlock(32,32);
  MMKernel<<<dimGrid,dimBlock>>>(A_d, B_d, C_d, Brow, Arow, Bcol);

  hipMemcpy(Cmatrix,C_d,size, hipMemcpyDeviceToHost);

  output_matrix(Cfile, Cmatrix, Arow, Bcol);
  
  //print_matrix(Cmatrix, Arow, Bcol);
  
  /* Free Stuff */
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  free(Amatrix);
  free(Bmatrix);
  free(Cmatrix);
  free(Amapped);
  free(Bmapped);

  return 0;
}
